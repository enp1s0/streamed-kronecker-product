#include <iostream>
#include <hipblas.h>
#include <hipblas.h>
#include <chrono>

// A,Gの行列のサイズ N x N (デフォルト値)
const int DEF_N = 200;

// バッチサイズ
const int C = 30;

// 計算回数
const int CALC = 10;

// A kp B * vec(X)
void k(float *A,float *B,float *X,float *R,int N,hipblasHandle_t cublas){
	float one = 1.0f,zero = 0.0f;
	hipblasSgemm(
			cublas,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			N,N,N,
			&one,
			A,N,
			X,N,
			&zero,
			R,N
			);
	// Rがちゃんと計算されているかは知らない
	hipblasSgemm(
			cublas,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			N,N,N,
			&one,
			R,N,
			B,N,
			&zero,
			R,N
			);
}

// バッチで
void batched_k(float *A[C],float *B[C],float *X[C],float *R[C],int N,hipblasHandle_t cublas[C]){
	for(int i = 0;i < C;i++){
		k(A[i],B[i],X[i],R[i],N,cublas[i]);
	}
}

int main(int argc,char **argv){
	float *A[C];
	float *B[C];
	float *X[C];
	float *R[C];
	hipblasHandle_t cublas[C];
	hipStream_t stream[C];

	int N = DEF_N;

	if(argc > 1){
		N = std::stoi(argv[1]);
	}


	std::cout<<"行列サイズ : "<<N<<" x "<<N<<std::endl
		<<"バッチサイズ : "<<C<<std::endl
		<<"計算回数 : "<<CALC<<std::endl;

	// 初期化
	for(int i = 0;i < C;i++){
		hipMalloc((void**)&A[i],sizeof(float)*N*N);
		hipMalloc((void**)&B[i],sizeof(float)*N*N);
		hipMalloc((void**)&X[i],sizeof(float)*N*N);
		hipMalloc((void**)&R[i],sizeof(float)*N*N);
		// cublasの用意とstreamの接続
		hipStreamCreate(stream+i);
		hipblasCreate( cublas+i );
		hipblasSetStream( cublas[i], stream[i]);
	}
	// ウォームアップ
	batched_k(A,B,X,R,N,cublas);
#ifdef BATCHED
		auto start = std::chrono::system_clock::now();
		for(int i = 0;i < CALC;i++)batched_k(A,B,X,R,N,cublas);
		hipDeviceSynchronize();
		auto stop = std::chrono::system_clock::now();
		std::cout<<"計算時間 : "<<std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count()/static_cast<float>(CALC)<<" [ms]"<<std::endl;
#else	
		hipblasHandle_t cub;
		hipblasCreate( &cub );
		auto start = std::chrono::system_clock::now();
		for(int j = 0;j < CALC;j++){
			for(int i = 0;i < C;i++){
				k(A[i],B[i],X[i],R[i],N,cub);
			}
		}
		hipDeviceSynchronize();
		auto stop = std::chrono::system_clock::now();
		std::cout<<"計算時間 : "<<std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count()/static_cast<float>(CALC)<<" [ms]"<<std::endl;
		hipblasDestroy( cub );
#endif

	for(int i = 0;i < C;i++){
		hipFree(A[i]);
		hipFree(B[i]);
		hipFree(X[i]);
		hipFree(R[i]);
		hipblasDestroy( cublas[i] );
		hipStreamDestroy( stream[i] );
	}

}
