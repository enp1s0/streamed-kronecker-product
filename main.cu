#include <iostream>
#include <hipblas.h>
#include <hipblas.h>
#include <chrono>

// A,Gの行列のサイズ N x N
const int N = 100;

// バッチサイズ
const int C = 5000;

// 計算回数
const int CALC = 100;

// A kp B * vec(X)
void k(float *A,float *B,float *X,float *R,hipblasHandle_t cublas){
	float one = 1.0f,zero = 0.0f;
	hipblasSgemm(
			cublas,
			HIPBLAS_OP_T,
			HIPBLAS_OP_N,
			N,N,N,
			&one,
			A,N,
			X,N,
			&zero,
			R,N
			);
	hipblasSgemm(
			cublas,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
			N,N,N,
			&one,
			R,N,
			B,N,
			&zero,
			R,N
			);
}

// バッチで
void batched_k(float *A[C],float *B[C],float *X[C],float *R[C],hipblasHandle_t cublas[C]){
	for(int i = 0;i < C;i++){
		k(A[i],B[i],X[i],R[i],cublas[i]);
	}
}

int main(){
	float *A[C];
	float *B[C];
	float *X[C];
	float *R[C];
	hipblasHandle_t cublas[C];
	hipStream_t stream[C];

	std::cout<<"行列サイズ : "<<N<<" x "<<N<<std::endl
		<<"バッチサイズ : "<<C<<std::endl
		<<"計算回数 : "<<CALC<<std::endl;

	// 初期化
	for(int i = 0;i < C;i++){
		hipMalloc((void**)&A[i],sizeof(float)*N*N);
		hipMalloc((void**)&B[i],sizeof(float)*N*N);
		hipMalloc((void**)&X[i],sizeof(float)*N*N);
		hipMalloc((void**)&R[i],sizeof(float)*N*N);
		// cublasの用意とstreamの接続
		hipStreamCreate(stream+i);
		hipblasCreate( cublas+i );
		hipblasSetStream( cublas[i], stream[i]);
	}
	// ウォームアップ
	batched_k(A,B,X,R,cublas);
#ifdef BATCHED
		auto start = std::chrono::system_clock::now();
		for(int i = 0;i < CALC;i++)batched_k(A,B,X,R,cublas);
		hipDeviceSynchronize();
		auto stop = std::chrono::system_clock::now();
		std::cout<<std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count()/static_cast<float>(CALC)<<"[ms]"<<std::endl;
#else	
		hipblasHandle_t cub;
		hipblasCreate( &cub );
		auto start = std::chrono::system_clock::now();
		for(int j = 0;j < CALC;j++){
			for(int i = 0;i < C;i++){
				k(A[i],B[i],X[i],R[i],cub);
			}
		}
		hipDeviceSynchronize();
		auto stop = std::chrono::system_clock::now();
		std::cout<<"seq "<<std::chrono::duration_cast<std::chrono::milliseconds>(stop-start).count()/static_cast<float>(CALC)<<"[ms]"<<std::endl;
		hipblasDestroy( cub );
#endif

	for(int i = 0;i < C;i++){
		hipFree(A[i]);
		hipFree(B[i]);
		hipFree(X[i]);
		hipFree(R[i]);
		hipblasDestroy( cublas[i] );
		hipStreamDestroy( stream[i] );
	}

}
